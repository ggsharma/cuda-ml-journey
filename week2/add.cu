
#include <hip/hip_runtime.h>

__global__ void add(int*a, int *b , int*c, size_t N){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while(tid < N){
        c[tid] = a[tid] + b[tid];
        tid += blockDim.x * gridDim.x;
    }
}
